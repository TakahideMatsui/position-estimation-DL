#include "hip/hip_runtime.h"

#include "BLSOM.h"
#include "SelectGPU.h"
#include<algorithm>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

#ifndef DIST
#define DIST(bx,by,x,y) ((bx-x)*(bx-x)+(by-y)*(by-y))
#endif // !DIST


#ifndef MAX
#define MAX( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#define CHECK(call)														\
{																		\
	const hipError_t error = call;										\
	if(error!=hipSuccess){												\
		printf("Error %s:%d \t",__FILE__,__LINE__);						\
		printf("code:%d, reason:%s\n",error,hipGetErrorString(error));	\
		exit(1);														\
	}																	\
}


bool checkAllocatedMemory(void* pointer) {
	if (pointer != NULL) {
		return true;
	}
	else {
		return false;
	}
}

BLSOM::BLSOM(int vec_dim, int map_width) :iAlfa(0.5), iBeta(40), t_alfa(30), t_beta(20),
										  vec_dim(vec_dim), map_width(map_width), flg_gpu(true), flg_iniBatch(false) {
	int device;
	
	this->map_height = 0;
	CHECK(SelectBestGPU(&device));

	if (flg_gpu) {
		CHECK(hipSetDevice(device));
	}
}

BLSOM::BLSOM(int vec_dim, int map_width, int map_height) :iAlfa(0.5), iBeta(40), t_alfa(30), t_beta(20),
														  vec_dim(vec_dim), map_width(map_width), map_height(map_height), flg_gpu(true), flg_iniBatch(false) {
	int device;
	CHECK(SelectBestGPU(&device));

	if (flg_gpu) {
		CHECK(hipSetDevice(device));
	}
}

BLSOM::BLSOM(int vec_dim, int map_width, int map_height,int device):iAlfa(0.5), iBeta(40), t_alfa(30), t_beta(20), 
																    vec_dim(vec_dim), map_width(map_width), map_height(map_height),flg_gpu(true), flg_iniBatch(false) {
	if (flg_gpu) {
		CHECK(hipSetDevice(device));
	}
}

BLSOM::BLSOM(int vec_dim, int map_width, int map_height, int device, int gpuFlag) : iAlfa(0.5), iBeta(40), t_alfa(30), t_beta(20), 
																					vec_dim(vec_dim),map_width(map_width),map_height(map_height),flg_gpu(gpuFlag), flg_iniBatch(false) {
	
	if (gpuFlag) {
		CHECK(hipSetDevice(device));
	}
}

BLSOM::~BLSOM() {
	
}

void BLSOM::Init(const float sdev1, const float sdev2, const float* rot1, const float* rot2, const float *aveVec) {

	if (map_height == 0) {
		this->map_height = (sdev2 / sdev1)*this->map_width;
	}

	if (flg_gpu) {
		
		this->d_mapWeight = thrust::device_vector<float>(map_width*map_height*vec_dim);
		this->d_weightS = thrust::device_vector<float>(map_width*map_height* (vec_dim));
		this->d_cntWeightS = thrust::device_vector<float>(map_width*map_height);
		this->d_node = thrust::device_vector<float>(map_width*map_height);
		this->d_rot1 = thrust::device_vector<float>(vec_dim);
		this->d_rot2 = thrust::device_vector<float>(vec_dim);
		this->d_aveVec = thrust::device_vector<float>(vec_dim);
		this->d_sdev = thrust::device_vector<float>(2);
		this->d_bmuPos = thrust::device_vector<int>(2);

		hipMemcpy(thrust::raw_pointer_cast(this->d_rot1.data()), rot1, this->vec_dim * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(thrust::raw_pointer_cast(this->d_rot2.data()), rot2, this->vec_dim * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(thrust::raw_pointer_cast(this->d_aveVec.data()), aveVec, this->vec_dim * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(thrust::raw_pointer_cast(this->d_sdev.data()), &sdev1, sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(thrust::raw_pointer_cast(this->d_sdev.data()+1), &sdev2, sizeof(float), hipMemcpyHostToDevice);
	}

	this->h_mapWeight = thrust::host_vector<float>(this->map_width*this->map_height*this->vec_dim);
	this->h_weightS = thrust::host_vector<float>(this->map_width*this->map_height* (this->vec_dim));
	this->h_cntWeightS = thrust::device_vector<float>(map_width*map_height);
	this->h_node = thrust::host_vector<float>(this->map_width*this->map_height);
	this->h_rot1 = thrust::host_vector<float>(this->vec_dim);
	this->h_rot2 = thrust::host_vector<float>(this->vec_dim);
	this->h_aveVec = thrust::host_vector<float>(this->vec_dim);
	this->h_sdev = thrust::host_vector<float>(2);
	this->h_bmuPos = thrust::host_vector<int>(2);

	memcpy(thrust::raw_pointer_cast(this->h_rot1.data()), rot1, this->vec_dim * sizeof(float));
	memcpy(thrust::raw_pointer_cast(this->h_rot2.data()), rot2, this->vec_dim * sizeof(float));
	memcpy(thrust::raw_pointer_cast(this->h_aveVec.data()), aveVec, this->vec_dim * sizeof(float));
	memcpy(thrust::raw_pointer_cast(this->h_sdev.data()), &sdev1, sizeof(float));
	memcpy(thrust::raw_pointer_cast(this->h_sdev.data()+1), &sdev2, sizeof(float));

	this->flg_iniBatch = true;
}

void BLSOM::SetTrainingData(const std::vector<std::vector<float>> train) {
	float* tempTrain;
	float* temp_begin;

	this->train_num = train.size();
	this->vec_dim = train[0].size();

	tempTrain = new float[train_num*this->vec_dim];
	temp_begin = tempTrain;

	thrust::host_vector<float> h_train = thrust::host_vector<float>(train_num*this->vec_dim);
	thrust::device_vector<float> d_train = thrust::device_vector<float>(train_num*this->vec_dim);

	for_each(train.begin(), train.end(), [&](std::vector<float> data) {memcpy(tempTrain, data.data(), data.size() * sizeof(float)); tempTrain += data.size();});

	memcpy(thrust::raw_pointer_cast(h_train.data()), temp_begin,train_num*this->vec_dim * sizeof(float));
	hipMemcpy(thrust::raw_pointer_cast(d_train.data()), thrust::raw_pointer_cast(h_train.data()), train_num*this->vec_dim * sizeof(float), hipMemcpyHostToDevice);
	this->d_trains.push_back(d_train);

	free(temp_begin);

}

void BLSOM::SetTrainingData(const std::vector<std::vector<std::vector<float>>> train) {
	
	for (auto trains: train) {
		this->SetTrainingData(trains);
	}
}


void BLSOM::check_mapWeight() {
	hipMemcpy(thrust::raw_pointer_cast(this->h_mapWeight.data()), thrust::raw_pointer_cast(this->d_mapWeight.data()), sizeof(float)*this->map_width*this->map_height*this->vec_dim, hipMemcpyDeviceToHost);

	for (int idy = 0; idy < map_height; idy++) {
		for (int idx = 0; idx < map_width; idx++) {
			//printf("%d %d \n",idy, idx);
			printf("%d", map_width*idy + idx);
			//printf("%d", map_width*vec_dim*idy + vec_dim*idx);
			/*
			for (int idz = 0; idz < vec_dim; idz++) {
				printf("%d :", map_width*vec_dim*idy + vec_dim*idx + idz);
				printf("%f ", this->h_mapWeight[map_width*vec_dim*idy + vec_dim*idx + idz]);
				printf("\n");
			}*/
			printf("\n");
		}
	}
}

__global__ void InitMapWeightFromGPU(float* mapWeight,float* ave_vec, float* sdev, float* rot1, float* rot2, const int map_width, const int map_height, const int vec_dim) {
	int ix = blockIdx.x*blockDim.x;
	int iy = blockIdx.y*blockDim.y;
	int idx = map_width*vec_dim*iy + vec_dim*ix + threadIdx.z;

	float sigmaB1 = 5 * sdev[0] * rot1[threadIdx.z];
	float sigmaB2 = 5 * sdev[1] * rot2[threadIdx.z];

	mapWeight[idx] = ave_vec[threadIdx.z]+ sigmaB1*((ix - (map_width / 2.0)) / map_width) + sigmaB2*((iy - (map_height / 2.0)) / map_height);
}

__global__ void setup_kernel(hiprandState *state, const int map_width, const int vec_dim)
{
	int ix = blockIdx.x*blockDim.x;
	int iy = blockIdx.y*blockDim.y;
	int idx = map_width*vec_dim*iy + vec_dim*ix + threadIdx.z;
	hiprand_init(1234, idx, 0, &state[idx]);
}

__global__ void InitMapWeightRandFromGPU(hiprandState* state, float* mapWeight, const int map_width, const int vec_dim, unsigned int l, unsigned int h) {
	unsigned int x;
	int ix = blockIdx.x*blockDim.x;
	int iy = blockIdx.y*blockDim.y;
	int idx = map_width*vec_dim*iy + vec_dim*ix + threadIdx.z;

	hiprandState  localState = state[idx];
	x = hiprand(&localState);

	state[idx] = localState;
	mapWeight[idx] = l + (x % (h+1));
}

void BLSOM::InitMapWeightRand() {

	dim3 block(1, 1, this->vec_dim);
	dim3 grid(this->map_width, this->map_height);
	thrust::device_vector<hiprandState> devStates(this->map_width * this->map_height * this->vec_dim);

	setup_kernel <<< grid, block >>> (thrust::raw_pointer_cast(devStates.data()),this->map_width, this->vec_dim);
	InitMapWeightRandFromGPU <<< grid, block >>>(thrust::raw_pointer_cast(devStates.data()),
		thrust::raw_pointer_cast(this->d_mapWeight.data()),
		this->map_width,
		this->vec_dim,
		0,
		255);
}

void BLSOM::InitMapWeightBatch() {
	dim3 block(1, 1, this->vec_dim);
	dim3 grid(this->map_width, this->map_height);

	if (this->flg_iniBatch) {
		InitMapWeightFromGPU << < grid, block >> > (thrust::raw_pointer_cast(this->d_mapWeight.data()),
													thrust::raw_pointer_cast(this->d_aveVec.data()),
													thrust::raw_pointer_cast(this->d_sdev.data()),
													thrust::raw_pointer_cast(this->d_rot1.data()),
													thrust::raw_pointer_cast(this->d_rot2.data()),
													this->map_width,
													this->map_height,
													this->vec_dim);
	}
	else {
		std::cerr << "Please call BLSOM::Init, before call InitMapWeightBatch." << std::endl;
	}
}

void BLSOM::InitMapWeight(int mode) {
	

	switch (mode){
	case INIT_BATCH:
		InitMapWeightBatch();
		break;

	case INIT_RANDOM:
		InitMapWeightRand();
		break;

	default:
		break;
	}
	
}

__global__ void InitNodeFromGPU(float* node,const int map_width) {
	int ix = blockIdx.x*blockDim.x;
	int iy = blockIdx.y*blockDim.y;
	int idx = map_width*iy + ix;
	node[idx] = 0;
}

__global__ void BMUFromGPU(float* input_xk, float* node, float* mapWeight, const int map_width, const int vec_dim) {
	int ix = blockIdx.x*blockDim.x;
	int iy = blockIdx.y*blockDim.y;
	int node_idx = map_width*iy + ix;
	int map_idx = map_width*vec_dim*iy + vec_dim*ix;// + threadIdx.z;
	
	for(int dim=0;dim<vec_dim;dim++)
		node[node_idx] += (mapWeight[map_idx+dim]-input_xk[dim])*(mapWeight[map_idx + dim] - input_xk[dim]);
	
}

int BLSOM::getBMUIndex() {
	thrust::device_vector<float>::iterator bgn_itr = d_node.begin();
	thrust::device_vector<float>::iterator bmu_itr = thrust::min_element(thrust::device, d_node.begin(), d_node.end());
	return thrust::distance(bgn_itr, bmu_itr);
}

void BLSOM::setBMUPosition() {
	int bmu_index = getBMUIndex();
	this->h_bmuPos[0] = bmu_index % (this->map_width);	//x���W�v�Z
	this->h_bmuPos[1] = bmu_index / (this->map_width);	//y���W�v�Z
	this->d_bmuPos = this->h_bmuPos;
}

__global__ void CalcWeightSFromGPU(float* input_xk, int* bmuPos, float* weightS,float* cntWeightS,
								   const int map_width, const int vec_dim,
								   const double iBeta, const double tBeta, const int lnum) {

	int ix = blockIdx.x*blockDim.x;
	int iy = blockIdx.y*blockDim.y;
	int weiS_idx = map_width*vec_dim*iy + vec_dim*ix;// +threadIdx.z;
	int cntS_idx = map_width*iy + ix;				//cntWeightS[ix][iy]


	float dist = DIST(bmuPos[0], bmuPos[1], ix, iy);
	float Beta = MAX(0, (iBeta*(1 - (lnum / tBeta))));

	if ((Beta*Beta - dist) >= 0) {
		//printf("calsWeightS\n");
		for (int dim = 0; dim < vec_dim; dim++) {
			weightS[weiS_idx + dim] += input_xk[dim];
		}
		cntWeightS[cntS_idx]++;
	}

}

__global__ void UpdateMapWeightFromGPU(float* mapWeight, float* weightS, float* cntWeightS,
									   const int map_width, const int vec_dim,
									   const double iAlfa, const double tAlfa, const int lnum) {
	int ix = blockIdx.x*blockDim.x;
	int iy = blockIdx.y*blockDim.y;
	int map_idx = map_width*vec_dim*iy + vec_dim*ix;// +threadIdx.z;
	int cntS_idx = map_width*iy + ix;							//weightS[ix][iy][vec_dim]

	float alfaFunc = MAX(0.01, (iAlfa*(1.0 - (lnum / tAlfa))));

	if (cntWeightS[cntS_idx] > 0) {
		for (int dim = 0; dim < vec_dim; dim++) {
			weightS[map_idx + dim] /= cntWeightS[cntS_idx];
			weightS[map_idx + dim] -= mapWeight[map_idx + dim];
			weightS[map_idx + dim] *= alfaFunc;
			mapWeight[map_idx + dim] += weightS[map_idx + dim];
		}
	}
}

void BLSOM::BMU(float* input_xk) {
	dim3 block(1, 1, this->vec_dim);
	dim3 grid(this->map_width, this->map_height);

	InitNodeFromGPU <<< grid, 1 >>> (thrust::raw_pointer_cast(this->d_node.data()),this->map_width);
	BMUFromGPU <<< grid,1 >>>(input_xk, thrust::raw_pointer_cast(this->d_node.data()), thrust::raw_pointer_cast(this->d_mapWeight.data()), this->map_width, this->vec_dim);
	setBMUPosition();
	
}

void BLSOM::CalcWeightS(float* input_xk, int Lnum) {
	dim3 block(1, 1, this->vec_dim);
	dim3 grid(this->map_width, this->map_height);

	CalcWeightSFromGPU <<<grid, 1 >>> (input_xk,
										   thrust::raw_pointer_cast(this->d_bmuPos.data()),
										   thrust::raw_pointer_cast(this->d_weightS.data()),
										   thrust::raw_pointer_cast(this->d_cntWeightS.data()),			   
										   this->map_width,
										   this->vec_dim,
										   this->iBeta,
										   this->t_beta,
										   Lnum);
											
}

void BLSOM::UpdateMapWeight(int Lnum) {
	dim3 block(1, 1, this->vec_dim);
	dim3 grid(this->map_width, this->map_height);

	UpdateMapWeightFromGPU <<<grid,1>>> (thrust::raw_pointer_cast(this->d_mapWeight.data()),
											 thrust::raw_pointer_cast(this->d_weightS.data()),
											 thrust::raw_pointer_cast(this->d_cntWeightS.data()),
											 
											 this->map_width,
											 this->vec_dim,
											 this->iAlfa,
											 this->t_alfa,
											 Lnum);
}

__global__ void InitCntWeightSFromGPU(float* cntWeightS) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	cntWeightS[idx] = 0;

}

__global__ void InitWeighSFromGPU(float* weightS) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	weightS[idx] = 0;
}

void BLSOM::Learning(int Lnum) {
	std::cout << "Learning Start" << std::endl;
	
	int nWeightS = this->map_width*this->map_height*this->vec_dim;
	dim3 weightS_block(this->vec_dim);
	dim3 weightS_grid((nWeightS+weightS_block.x-1)/weightS_block.x);

	int nCntWeightS = this->map_width*this->map_height;
	dim3 cntWeightS_block(this->map_height);
	dim3 cntWeightS_grid((nCntWeightS+cntWeightS_block.x-1)/cntWeightS_block.x);


	for (int l = 0; l < Lnum; l++) {
		std::cout << "Learning : " << l << "/" << Lnum << "\r";

		for (int i = 0; i < this->d_trains.size(); i++) {
			
			InitCntWeightSFromGPU << <cntWeightS_grid, cntWeightS_block >> > (thrust::raw_pointer_cast(this->d_cntWeightS.data()));
			InitWeighSFromGPU << <weightS_grid, weightS_block >> > (thrust::raw_pointer_cast(this->d_weightS.data()));

			for (int j = 0; j < this->train_num; j++) {
				this->BMU(thrust::raw_pointer_cast(&(this->d_trains[i][j*(this->vec_dim)]))); //�Y�������C��
				this->CalcWeightS(thrust::raw_pointer_cast(&(this->d_trains[i][j*(this->vec_dim)])), l);
			}
			this->UpdateMapWeight(l);
		}
	}

	std::cout << "Learning Finish" << std::endl;
}

float* BLSOM::GetSOMMap() {
	this->h_mapWeight = this->d_mapWeight;
	return thrust::raw_pointer_cast(this->h_mapWeight.data());
}

void BLSOM::d_showWeightS() {
	for (int h = 0; h < this->map_height; h++) {
		for (int w = 0; w < this->map_width; w++) {
			std::cout << "(" << w << "," << h << "): ";
			for (int d = 0; d < this->vec_dim; d++) {
				 std::cout << this->d_weightS[h*map_width*vec_dim + w*vec_dim + d] << " ";
			}
			std::cout << "\n";
		}
	}
}

void BLSOM::d_showMapWeight() {
	for (int h = 0; h < this->map_height; h++) {
		for (int w = 0; w < this->map_width; w++) {
			std::cout << "(" << w << "," << h << "): ";
			for (int d = 0; d < this->vec_dim; d++) {
				std::cout << this->d_mapWeight[h*map_width*vec_dim + w*vec_dim + d] << " ";
			}
			std::cout << "\n";
		}
	}
}