#include"SelectGPU.h"

hipError_t SelectBestGPU(int* device) {
	hipError_t err;
	int device_count;

	err = hipGetDeviceCount(&device_count);

	if (err != hipSuccess) {
		fprintf_s(stderr, "Cannot get device count.");
		return err;
	}

	int *device_ids;

	device_ids = (int*)malloc(sizeof(int) * device_count);

	hipDeviceProp_t device_prop;
	int maxClock = 0;
	for (int id = 0; id < device_count; id++) {
		hipGetDeviceProperties(&device_prop, id);
		if (maxClock < device_prop.clockRate) {
			maxClock = device_prop.clockRate;
			*device = id;
		}
	}
	return hipSuccess;
}

hipError_t SelectGPU(int* device) {
	hipError_t err;
	int device_count;

	err = hipGetDeviceCount(&device_count);

	if (err != hipSuccess) {
		fprintf_s(stderr, "Cannot get device count.");
		return err;
	}

	int *device_ids;

	device_ids = (int*)malloc(sizeof(int) * device_count);

	hipDeviceProp_t device_prop;

	printf("please chose GPU for Below.\n\n");
	for (int id = 0; id < device_count; id++) {
		hipGetDeviceProperties(&device_prop, id);
		device_ids[id] = id;
		printf("[%d]:%s\n", id, device_prop.name);
	}

	int id;
	printf("please input id >>");
	scanf("%d", &id);

	*device = device_ids[id];
	return hipSuccess;
}

hipError_t SelectGPU(int *device, const char* strDevice) {
	hipError_t err;
	int device_count;

	err = hipGetDeviceCount(&device_count);

	if (err != hipSuccess) {
		fprintf_s(stderr, "Cannot get device count.");
		return err;
	}

	int *device_ids;

	device_ids = (int*)malloc(sizeof(int) * device_count);
	hipDeviceProp_t device_prop;

	int hits = 0;
	for (int id = 0; id < device_count; id++) {
		hipGetDeviceProperties(&device_prop, id);
		if (strcmp(device_prop.name, strDevice) == 0) {
			*device = id;
			return hipSuccess;
		}
	}
	return hipErrorInvalidDevice;
}

hipError_t SelectGPU(int* device, const std::string strDevice) {
	hipError_t err;
	int device_count;

	err = hipGetDeviceCount(&device_count);

	if (err != hipSuccess) {
		fprintf_s(stderr, "Cannot get device count.");
		return err;
	}

	int *device_ids;

	device_ids = (int*)malloc(sizeof(int) * device_count);
	hipDeviceProp_t device_prop;

	int hits = 0;
	for (int id = 0; id < device_count; id++) {
		hipGetDeviceProperties(&device_prop, id);
		if (strcmp(device_prop.name, strDevice.c_str()) == 0) {
			*device = id;
			return hipSuccess;
		}
	}
	return hipErrorInvalidDevice;


}